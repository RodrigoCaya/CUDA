#include <iostream>
#include <time.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include <fstream>
#include <string>
#include <sys/types.h>
using namespace std;

#define BS 256
#define X 3
#define Y 3
#define cantStream 6
#define PYTHON_COMMAND 0 // 0 si el comando de python NO lleva el 3. cualquier otro valor si lo lleva.

//
// Funcion que ejecuta el comando de python para convertir la imagen en formato txt
// en formato txt a PNG.
// Hay que cambiar PYTHON_COMMAND dependiendo de como se ejecuta el comando en el PC.
//
int TXTtoRGB(){
    if (PYTHON_COMMAND == 0){
        cout << "Reconstruyendo Imagen..." << endl;
        system("python TXTtoRGB.py");
        return 1;
    }
    else if(system("python3 TXTtoRGB.py")){
        cout << "Reconstruyendo Imagen..." << endl;
        return 1;
    }
    else{
        cout << "Error al convertir el txt a imagen" << endl;
        return 0;
    }
}

// Funcion que ejecuta el comando de python para reconstruir la imagen en formato PNG
// a formato txt.
// Hay que cambiar PYTHON_COMMAND dependiendo de como se ejecuta el comando en el PC.

int RGBtoTXT(){
    if (PYTHON_COMMAND == 0){
        cout << "Transformando Imagen..." << endl;
        system("python IMGtoTXT.py < name.txt");
        return 1;
    }
    else if(system("python3 IMGtoTXT.py < name.txt")){
        cout << "Transformando Imagen..." << endl;
        return 1;
    }
    else{
        cout << "Error al transformar el txt a imagen" << endl;
        return 0;
    }
}

//
// Funcion que imprime el ultimo error arrojado por CUDA
//
void cudaCheckError(int i) {
    hipError_t e=hipGetLastError();
    if(e!=hipSuccess) {
        printf("%d.- Cuda failure %s:%d: '%s'\n", i,__FILE__,__LINE__,hipGetErrorString(e));
        exit(0);
    }
}

/*
 *  Escritura Archivo txt
    Funcion extraida de actividad de curso
 */
void Write(float* R, float* G, float* B, 
	       int M, int N, const char *filename) {
    FILE *fp;
    fp = fopen(filename, "w");
    fprintf(fp, "%d %d\n", M, N);
    for(int i = 0; i < M*N-1; i++)
        fprintf(fp, "%f ", R[i]);
    fprintf(fp, "%f\n", R[M*N-1]);
    for(int i = 0; i < M*N-1; i++)
        fprintf(fp, "%f ", G[i]);
    fprintf(fp, "%f\n", G[M*N-1]);
    for(int i = 0; i < M*N-1; i++)
        fprintf(fp, "%f ", B[i]);
    fprintf(fp, "%f\n", B[M*N-1]);
    fclose(fp);
}

/*
 *  Lectura Archivo txt
 */
void Read(float** R, float** G, float** B, int *M, int *N, const char *filename) {    
    FILE *fp;
    fp = fopen(filename, "r");
    fscanf(fp, "%d %d\n", M, N);

    int imsize = (*M) * (*N);
    float* R1 = new float[imsize];
    float* G1 = new float[imsize];
    float* B1 = new float[imsize];

    for(int i = 0; i < imsize; i++)
        fscanf(fp, "%f ", &(R1[i]));
    for(int i = 0; i < imsize; i++)
        fscanf(fp, "%f ", &(G1[i]));
    for(int i = 0; i < imsize; i++)
        fscanf(fp, "%f ", &(B1[i]));
    
    fclose(fp);
    *R = R1; *G = G1; *B = B1;
}

__global__ void kernelStream(float* R, float* Rx, float *Ry, int M, int N, int Mout, int Nout, int *k1, int *k2, int tam){
    int tid = threadIdx.x + blockDim.x * blockIdx.x;// 
	if (tid < tam){
        float v1 = 0, v2 = 0;
        int fila = tid + (tid/Nout)*2;
        for(int i = 0; i<Y ; i++){
            for(int j = 0; j<X ; j++){
                v1 += R[j+i*N+fila]*k1[j+i*Y];
                v2 += R[j+i*N+fila]*k2[j+i*Y];
            }
        }
        Rx[tid] = v1;
        Ry[tid] = v2;
    }
}


__global__ void kernelConvolucion(float* R, float* Rx, float *Ry, int M, int N, int Mout, int Nout, int *k1, int *k2){
	int tid = threadIdx.x + blockDim.x * blockIdx.x;
	if (tid < Mout*Nout){
        float v1 = 0, v2 = 0;
        int fila = tid + (tid/Nout)*2;
        for(int i = 0; i<Y ; i++){
            for(int j = 0; j<X ; j++){
                v1 += R[j+i*N+fila]*k1[j+i*Y];
                v2 += R[j+i*N+fila]*k2[j+i*Y];
            }
        }
        Rx[tid] = v1;
        Ry[tid] = v2;
    }
}


__global__ void kernelFila(float* R, float* Rx, float *Ry, int M, int N, int Mout, int Nout, int *k1, int *k2){
	int tid = threadIdx.x + blockDim.x * blockIdx.x;
	if (tid < Mout){
        float v1, v2;
        for(int k=0; k<Nout ; k++){
            v1 = 0;
            v2 = 0;
            for(int i = 0; i<Y ; i++){
                for(int j = 0; j<X ; j++){
                    v1 += R[j+k+i*N+tid*N]*k1[j+i*Y];
                    v2 += R[j+k+i*N+tid*N]*k2[j+i*Y];
                }
            }
            Rx[k+tid*Nout] = v1;
            Ry[k+tid*Nout] = v2;
        }
    }
}

void kernelCPU(float* R, float* Rx, float *Ry, int M, int N, int Mout, int Nout, int *k1, int *k2){
    float v1, v2;
    for(int h=0 ; h<Mout ; h++){
        for(int k=0; k<Nout ; k++){
            v1 = 0;
            v2 = 0;
            for(int i = 0; i<Y ; i++){
                for(int j = 0; j<X ; j++){
                    v1 += R[j+k+i*N+h*N]*k1[j+i*Y];
                    v2 += R[j+k+i*N+h*N]*k2[j+i*Y];
                }
            }
            Rx[k+h*Nout] = v1;
            Ry[k+h*Nout] = v2;
        }
    }
}

void blancoynegro(float* R, float* G, float* B, int M, int N){
    float prom = 0;
    int imsize = (M) * (N);
    for(int i = 0; i < imsize; i++){
        prom = (R[i]+G[i]+B[i] )/ 3;
        (R[i]) = prom;
        (G[i]) = prom;
        (B[i]) = prom;
    }
}

void copiar(float *Rhost, float *Ghost, float *Bhost, float *Rxhost, float *Ryhost, int Mout, int Nout){
    float tmp;
    //norma
    for(int i = 0; i < Mout*Nout; i++){
        tmp = sqrt( (pow(Rxhost[i], 2)+ pow(Ryhost[i], 2)) );
        if(tmp > 1) tmp = 1;
        Rhost[i] = tmp;
        Ghost[i] = tmp;
        Bhost[i] = tmp;
    }

    Write(Rhost, Ghost, Bhost, Mout, Nout, "salida.txt");
}

void callKernelFila(float * Rhost, int N, int M, int Mout, int Nout, int * k1, int * k2){
    float *Rdev, *Rx, *Ry, *Rxhost, *Ryhost, dt;
    hipEvent_t ct1, ct2;
    int *k1dev, *k2dev;
    
    int GS = (int)ceil((float) Mout / BS);

    hipMalloc((void**)&Rdev, M * N * sizeof(float));
    //cudaMemcpy(Rdev, Rhost, M * N * sizeof(float), cudaMemcpyHostToDevice);
    hipMemcpy(Rdev, Rhost, M * N * sizeof(float), hipMemcpyHostToDevice);
    cudaCheckError(1);

    hipMalloc((void**)&k1dev, X * Y * sizeof(int));
    hipMalloc((void**)&k2dev, X * Y * sizeof(int));
    hipMemcpy(k1dev, k1, X * Y * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(k2dev, k2, X * Y * sizeof(int), hipMemcpyHostToDevice);
    cudaCheckError(2);

    hipMalloc((void**)&Rx, Mout * Nout * sizeof(float));
    hipMalloc((void**)&Ry, Mout * Nout * sizeof(float));
    cudaCheckError(3);

    hipEventCreate(&ct1);
    hipEventCreate(&ct2);
    hipEventRecord(ct1);

    kernelFila<<<GS, BS>>>(Rdev, Rx, Ry, M, N, Mout, Nout, k1dev, k2dev);
    // cudaCheckError(4);

    Rxhost = new float[Mout*Nout];
	Ryhost = new float[Mout*Nout];
    hipMemcpy(Rxhost, Rx, Mout * Nout * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(Ryhost, Ry, Mout * Nout * sizeof(float), hipMemcpyDeviceToHost);
    
    hipEventRecord(ct2);
    hipEventSynchronize(ct2);
    hipEventElapsedTime(&dt, ct1, ct2);
    cout << "Tiempo GPU una hebra por fila: " << dt << "[ms]" << endl;
    // cudaCheckError(5);

    float *Rfinal= new float[Mout*Nout];
    float *Gfinal = new float[Mout*Nout];
    float *Bfinal = new float[Mout*Nout];
    copiar(Rfinal, Gfinal, Bfinal, Rxhost, Ryhost, Mout, Nout);

}

void callCPU(float * Rhost, int N, int M, int Mout, int Nout, int * k1, int * k2){
    clock_t t1, t2;
    double ms;
    t1 = clock();
    float *Rx = new float[Mout*Nout];
    float *Ry = new float[Mout*Nout];
    kernelCPU(Rhost, Rx, Ry, M, N, Mout, Nout, k1, k2);
    t2 = clock();
    ms = 1000.0 * (double)(t2 - t1) / CLOCKS_PER_SEC;
    cout << "Tiempo CPU es: "<< ms << "[ms]" << endl;

    float *Rfinal= new float[Mout*Nout];
    float *Gfinal = new float[Mout*Nout];
    float *Bfinal = new float[Mout*Nout];
    copiar(Rfinal, Gfinal, Bfinal, Rx, Ry, Mout, Nout);
}

void callKernelConv(float * Rhost, int N, int M, int Mout, int Nout, int * k1, int * k2){
    float *Rdev, *Rx, *Ry, *Rxhost, *Ryhost, dt;
    hipEvent_t ct1, ct2;
    int *k1dev, *k2dev;
    
    int GS = (int)ceil((float) Mout*Nout / BS);

    cudaCheckError(1);

    hipMalloc((void**)&k1dev, X * Y * sizeof(int));
    hipMalloc((void**)&k2dev, X * Y * sizeof(int));
    hipMemcpy(k1dev, k1, X * Y * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(k2dev, k2, X * Y * sizeof(int), hipMemcpyHostToDevice);
    cudaCheckError(2);

    hipMalloc((void**)&Rx, Mout * Nout * sizeof(float));
    hipMalloc((void**)&Ry, Mout * Nout * sizeof(float));
    cudaCheckError(3);
    hipMalloc((void**)&Rdev, M * N * sizeof(float));
    
    Rxhost = new float[Mout*Nout];
	Ryhost = new float[Mout*Nout];

    hipEventCreate(&ct1);
    hipEventCreate(&ct2);
    hipEventRecord(ct1);
    
    hipMemcpy(Rdev, Rhost, M * N * sizeof(float), hipMemcpyHostToDevice);

    kernelConvolucion<<<GS, BS>>>(Rdev, Rx, Ry, M, N, Mout, Nout, k1dev, k2dev);
    cudaCheckError(4);

    hipMemcpy(Rxhost, Rx, Mout * Nout * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(Ryhost, Ry, Mout * Nout * sizeof(float), hipMemcpyDeviceToHost);
    cudaCheckError(5);

    hipEventRecord(ct2);
    hipEventSynchronize(ct2);
    hipEventElapsedTime(&dt, ct1, ct2);
    cout << "Tiempo GPU una hebra por convolucion: " << dt << "[ms]" << endl;
    
    float *Rfinal= new float[Mout*Nout];
    float *Gfinal = new float[Mout*Nout];
    float *Bfinal = new float[Mout*Nout];
    copiar(Rfinal, Gfinal, Bfinal, Rxhost, Ryhost, Mout, Nout);
}

void callKernelStream(float * Rhost, int N, int M, int Mout, int Nout, int * k1, int * k2){
    hipStream_t streams[cantStream];
    float *Rdev, *Rxhost, *Ryhost, dt;
    float *RxStream1;
    float *RyStream1;
    hipEvent_t ct1, ct2;
    int *k1dev, *k2dev;

    hipHostMalloc((void **)&Rxhost, Mout * Nout * sizeof(float), hipHostMallocDefault);
    hipHostMalloc((void **)&Ryhost,  Mout * Nout * sizeof(float), hipHostMallocDefault);

    // int cantStream = 4;
    int GS = (int)ceil((float) (Mout/cantStream)*Nout / BS);
    int GS4 = (int)ceil((float) ((Mout+Mout%cantStream)/cantStream)*Nout / BS);

    int size = (int)(Mout/cantStream)*Nout;
    // int size4 = (int)((Mout/cantStream)+Mout%cantStream)*Nout;   

    hipMalloc((void **)&RxStream1, size*cantStream * sizeof(float));
    hipMalloc((void **)&RyStream1, size*cantStream * sizeof(float));
    
    // copiar el kernel de convolucion a memoria de gpu
    hipMalloc((void **)&k1dev, X * Y * sizeof(float));
    hipMalloc((void **)&k2dev, X * Y * sizeof(float));
    hipMemcpy(k1dev, k1, X * Y * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(k2dev, k2, X * Y * sizeof(float), hipMemcpyHostToDevice);
    
    float * RH;
    hipHostMalloc((void **)&RH, M * N * sizeof(float), hipHostMallocDefault); //para copiar a los streams de forma eficiente

    for(int i = 0; i < M*N; i++){
        RH[i] = Rhost[i];
    }

    int sizeFull = (int)(M/cantStream)*N;
    int sizeFullOut = (int)(Mout/cantStream)*Nout;
    
    hipMalloc((void **)&Rdev, (sizeFull)* cantStream * sizeof(float));

    hipEventCreate(&ct1);
    hipEventCreate(&ct2);
    hipEventRecord(ct1);

    for(int i = 0; i < cantStream; i++){
        hipStreamCreate(&streams[i]);
        hipMemcpyAsync(&Rdev[i*sizeFull], &RH[i*sizeFull], (sizeFull) * sizeof(float), hipMemcpyHostToDevice, streams[i]);
        kernelStream<<<GS, BS, 0, streams[i]>>>(Rdev+i*sizeFull, RxStream1+i*size, RyStream1+i*size, M, N, Mout, Nout, k1dev, k2dev, size);
        hipMemcpyAsync(&Rxhost[i*sizeFullOut], RxStream1+i*size, size*sizeof(float), hipMemcpyDeviceToHost, streams[i]);
        hipMemcpyAsync(&Ryhost[i*sizeFullOut], RyStream1+i*size, size*sizeof(float), hipMemcpyDeviceToHost, streams[i]);
    }

    hipEventRecord(ct2);
    hipEventSynchronize(ct2);
    hipEventElapsedTime(&dt, ct1, ct2);
	printf("Tiempo GPU Streams: %f[ms]\n", dt);

    hipDeviceSynchronize();


    float *Rfinal= new float[Mout*Nout];
    float *Gfinal = new float[Mout*Nout];
    float *Bfinal = new float[Mout*Nout];
    copiar(Rfinal, Gfinal, Bfinal, Rxhost, Ryhost, Mout, Nout);
    
}

int main(){

    //inicializacion
    //se convierte la imagen a blanco y negro
    float *Rhost, *Ghost, *Bhost;
    int M, N, Mout, Nout; //M filas, N columnas

    ofstream file;
    file.open("name.txt");

    string name;
    cout << "Ingrese el nombre de la foto sin la extension: ";
    cin >> name;
    file << name;
    file.close();
    RGBtoTXT();

    name.append(".txt");

    char Name[500];
    for(int i = 0; i < name.size(); i++){
        Name[i] = name[i];
    }
    Name[name.size()] = '\0';

    Read(&Rhost, &Ghost, &Bhost, &M, &N, Name); 
    blancoynegro(Rhost, Ghost, Bhost, M, N);
    Nout = N - 2;
	Mout = M - 2;
    
    int *k1{ new int[9]{ -1, 0, 1, -2, 0, 2, -1, 0, 1 } };
    int *k2{ new int[9]{ -1, -2, -1, 0, 0, 0, 1, 2, 1 } };

    // llamada a la implementacion de cpu
    callCPU(Rhost, N, M, Mout, Nout, k1, k2);
    TXTtoRGB();

    // llamada a la implementaci??n del kernel usando una hebra por fila. 
    callKernelFila(Rhost, N, M, Mout, Nout, k1, k2);
    TXTtoRGB();

    // llamada a la implementaci??n del kernel usando una hebra por fila. 
    callKernelConv(Rhost, N, M, Mout, Nout, k1, k2);
    TXTtoRGB();

    // llamada al kernel usando streams y un kernel por calculo.
    callKernelStream(Rhost, N, M, Mout, Nout, k1, k2);
    TXTtoRGB();

    return 0;
}