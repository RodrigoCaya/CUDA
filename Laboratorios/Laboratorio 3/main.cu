#include <time.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#define BS 256
/*
 *  Generador de matriz
 */
void genMatrix(int** matrix, int N, int M) {   
    int* matrix1 = new int[M*N];
    srand(1);
	for(int i = 0; i < M*N; i++)
		matrix1[i] = rand() % 1000 +1;
        // matrix1[i] = 1;
    *matrix = matrix1;
}

 /*
  *  Kernel inciso A
  */
__global__ void kernelA(int *A, int *x, int *b, int N){
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int xtid = tid/N; // 0 a N, +1 cada N
    int ytid = tid%N; // 0 a N cada N
	if (tid < N*N){
        atomicAdd(&b[xtid], A[tid]*x[ytid]);
	}
}

 /*
  *  Kernel inciso B
  */
__global__ void kernelx(int *A, int *x, int *b, int N){
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if (tid < N){
        for(int i = 0; i < N; i++){
            atomicAdd(&b[i], A[tid+i*N]*x[tid]);
        }
	}
}

 /*
  *  Kernel inciso C
  */
__global__ void kernelb(int *A, int *x, int *b, int N){
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if (tid < N){
        int total = 0;
        for(int i = 0; i < N; i++){
            total += A[i+tid*N]*x[i];
        }
        b[tid] = total;
	}
}

 /*
  *  Kernel inciso D
  */
__global__ void kernelRed(int *A, int *x, int *b, int N){
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < N){
        extern __shared__ int ax[];
        int size;
        for(int i = 0; i < N; i++){
            ax[threadIdx.x] = A[i*N + tid]*x[tid];
            __syncthreads();
            for (size = 256/2; size>0; size/=2) {
                if (threadIdx.x<size) atomicAdd(&ax[threadIdx.x], ax[threadIdx.x+size]);
                __syncthreads();
            }
            if (threadIdx.x == 0){
                atomicAdd(&b[i], ax[0]);
            }
        }
    }
}

/*
  *  Kernel inciso E
  */
__global__ void kernelSM(int *A,int *x, int *b, int N){
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    extern __shared__ int vx[];
	if (tid < N){
        int total = 0;
        for(int i = 0; i < N; i++){
            vx[threadIdx.x] = x[i];
            __syncthreads();
            total += A[tid*N + i]*vx[threadIdx.x];
            // __syncthreads();
        }
        atomicAdd(&b[tid], total);
	}
}

/*
  *  Kernel inciso F
  */
__constant__ int X[10000];
__global__ void kernelCM(int *A, int *b, int N){
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < N){
        int total = 0;
        for(int i = 0; i < N; i++){
            total += A[i+tid*N]*X[i];
        }
        b[tid] = total;
    }
}


int main(){
    //clock_t t1, t2;
	//double ms;
	hipEvent_t ct1, ct2;
	float dt;
    int *A, *x, *b;
    int *Ahost, *xhost, *bhost;
	int N = 10000, M = 10000;
    int gs, bs = 256;
    gs = (int)ceil((float)N*N / bs);

    genMatrix(&Ahost, N, M);
	genMatrix(&xhost, 1, N);
    
    //inciso a

    hipMalloc((void**)&A, N * M * sizeof(int));
    hipMalloc((void**)&x, N * sizeof(int));
    hipMemcpy(A, Ahost, N * M * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(x, xhost, N * sizeof(int), hipMemcpyHostToDevice);

    hipMalloc((void**)&b, M * sizeof(int));
	    
    hipEventCreate(&ct1);
    hipEventCreate(&ct2);
    hipEventRecord(ct1);
    kernelA<<<gs, bs>>>(A, x, b, N);
    hipEventRecord(ct2);
    hipEventSynchronize(ct2);
    hipEventElapsedTime(&dt, ct1, ct2);
	printf("a) Tiempo GPU: %f[ms]\n", dt);

	bhost = new int[M];
	hipMemcpy(bhost, b, M * sizeof(int), hipMemcpyDeviceToHost);
	delete[] bhost;
	hipFree(b); hipFree(A); hipFree(x);


    //inciso b

    gs = (int)ceil((float)N / bs);
    hipMalloc((void**)&A, N * M * sizeof(int));
    hipMalloc((void**)&x, N * sizeof(int));
    hipMemcpy(A, Ahost, N * M * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(x, xhost, N * sizeof(int), hipMemcpyHostToDevice);
    hipMalloc((void**)&b, M * sizeof(int));
	    
    hipEventCreate(&ct1);
    hipEventCreate(&ct2);
    hipEventRecord(ct1);
    kernelx<<<gs, bs>>>(A, x, b, N);
    hipEventRecord(ct2);
    hipEventSynchronize(ct2);
    hipEventElapsedTime(&dt, ct1, ct2);
	printf("b) Tiempo GPU: %f[ms]\n", dt);

	bhost = new int[M];
	hipMemcpy(bhost, b, M * sizeof(int), hipMemcpyDeviceToHost);
	delete[] bhost;
	hipFree(b); hipFree(A); hipFree(x);

    //inciso c

    hipMalloc((void**)&A, N * M * sizeof(int));
    hipMalloc((void**)&x, N * sizeof(int));
    hipMemcpy(A, Ahost, N * M * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(x, xhost, N * sizeof(int), hipMemcpyHostToDevice);
    hipMalloc((void**)&b, M * sizeof(int));
	    
    hipEventCreate(&ct1);
    hipEventCreate(&ct2);
    hipEventRecord(ct1);
    kernelb<<<gs, bs>>>(A, x, b, N);
    hipEventRecord(ct2);
    hipEventSynchronize(ct2);
    hipEventElapsedTime(&dt, ct1, ct2);
	printf("c) Tiempo GPU: %f[ms]\n", dt);

	bhost = new int[M];
	hipMemcpy(bhost, b, M * sizeof(int), hipMemcpyDeviceToHost);
	delete[] bhost;
	hipFree(b); hipFree(A); hipFree(x);

    //inciso D
    gs = (int)ceil((float)N / bs);
    hipMalloc((void**)&A, N * M * sizeof(int));
    hipMalloc((void**)&x, N * sizeof(int));
    hipMemcpy(A, Ahost, N * M * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(x, xhost, N * sizeof(int), hipMemcpyHostToDevice);
    hipMalloc((void**)&b, M * sizeof(int));
	hipMemset(b, 0, M);

    hipEventCreate(&ct1);
    hipEventCreate(&ct2);
    hipEventRecord(ct1);

    kernelRed<<<gs, bs, bs*sizeof(int)>>>(A, x, b, N);
    hipEventRecord(ct2);
    hipEventSynchronize(ct2);
    hipEventElapsedTime(&dt, ct1, ct2);
	printf("d) Tiempo GPU: %f[ms]\n", dt);

	bhost = new int[M]();
	hipMemcpy(bhost, b, M * sizeof(int), hipMemcpyDeviceToHost);
	delete[] bhost;
	hipFree(b); hipFree(A); hipFree(x);


    //inciso E
    gs = (int)ceil((float)N / bs);
    hipMalloc((void**)&A, N * M * sizeof(int));
    hipMalloc((void**)&x, N * sizeof(int));
    hipMemcpy(A, Ahost, N * M * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(x, xhost, N * sizeof(int), hipMemcpyHostToDevice);
    hipMalloc((void**)&b, M * sizeof(int));
	hipMemset(b, 0, M);

    hipEventCreate(&ct1);
    hipEventCreate(&ct2);
    hipEventRecord(ct1);
    kernelSM<<<gs, bs, bs*sizeof(int)>>>(A, x, b, N);
    hipEventRecord(ct2);
    hipEventSynchronize(ct2);
    hipEventElapsedTime(&dt, ct1, ct2);
	printf("e) Tiempo GPU: %f[ms]\n", dt);

	bhost = new int[M]();
	hipMemcpy(bhost, b, M * sizeof(int), hipMemcpyDeviceToHost);
	delete[] bhost;
	hipFree(b); hipFree(A); hipFree(x);

    //inciso F
    gs = (int)ceil((float)N / bs);
    hipMalloc((void**)&A, N * M * sizeof(int));
    hipMemcpy(A, Ahost, N * M * sizeof(int), hipMemcpyHostToDevice);
    hipMalloc((void**)&b, M * sizeof(int));
	hipMemset(b, 0, M);
    hipMemcpyToSymbol(HIP_SYMBOL(X), xhost, N*sizeof(int), 0, hipMemcpyHostToDevice);

    hipEventCreate(&ct1);
    hipEventCreate(&ct2);
    hipEventRecord(ct1);

    kernelCM<<<gs, bs>>>(A, b, N);
    hipEventRecord(ct2);
    hipEventSynchronize(ct2);
    hipEventElapsedTime(&dt, ct1, ct2);
	printf("f) Tiempo GPU: %f[ms]\n", dt);

	bhost = new int[M]();
	hipMemcpy(bhost, b, M * sizeof(int), hipMemcpyDeviceToHost);
	delete[] bhost;
	hipFree(b); hipFree(A); hipFree(x);

    return 0;
}